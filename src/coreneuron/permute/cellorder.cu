#include "hip/hip_runtime.h"
/*
# =============================================================================
# Copyright (c) 2016 - 2021 Blue Brain Project/EPFL
#
# See top-level LICENSE file for details.
# =============================================================================
*/

#include "coreneuron/utils/utils_cuda.h"
#include "coreneuron/permute/cellorder.hpp"
#include "coreneuron/network/tnode.hpp"
#include "coreneuron/sim/multicore.hpp"

namespace coreneuron {

__device__ void triang_interleaved2_device(NrnThread* nt,
                                           int icore,
                                           int ncycle,
                                           int* stride,
                                           int lastnode) {
    int icycle = ncycle - 1;
    int istride = stride[icycle];
    int i = lastnode - istride + icore;

    int ip;
    double p;
    while (icycle >= 0) {
        // most efficient if istride equal warpsize, else branch divergence!
        if (icore < istride) {
            ip = nt->_v_parent_index[i];
            p = nt->_actual_a[i] / nt->_actual_d[i];
            atomicAdd(&nt->_actual_d[ip], -p * nt->_actual_b[i]);
            atomicAdd(&nt->_actual_rhs[ip], -p * nt->_actual_rhs[i]);
        }
        --icycle;
        istride = stride[icycle];
        i -= istride;
    }
}

__device__ void bksub_interleaved2_device(NrnThread* nt,
                                          int root,
                                          int lastroot,
                                          int icore,
                                          int ncycle,
                                          int* stride,
                                          int firstnode) {
    for (int i = root; i < lastroot; i += warpsize) {
        nt->_actual_rhs[i] /= nt->_actual_d[i];  // the root
    }

    int i = firstnode + icore;

    int ip;
    for (int icycle = 0; icycle < ncycle; ++icycle) {
        int istride = stride[icycle];
        if (icore < istride) {
            ip = nt->_v_parent_index[i];
            nt->_actual_rhs[i] -= nt->_actual_b[i] * nt->_actual_rhs[ip];
            nt->_actual_rhs[i] /= nt->_actual_d[i];
        }
        i += istride;
    }
}

__global__ void solve_interleaved2_kernel(NrnThread* nt, InterleaveInfo* ii, int ncore) {
    int icore = blockDim.x * blockIdx.x + threadIdx.x;

    int* ncycles = ii->cellsize;         // nwarp of these
    int* stridedispl = ii->stridedispl;  // nwarp+1 of these
    int* strides = ii->stride;           // sum ncycles of these (bad since ncompart/warpsize)
    int* rootbegin = ii->firstnode;      // nwarp+1 of these
    int* nodebegin = ii->lastnode;       // nwarp+1 of these

    int iwarp = icore / warpsize;     // figure out the >> value
    int ic = icore & (warpsize - 1);  // figure out the & mask
    int ncycle = ncycles[iwarp];
    int* stride = strides + stridedispl[iwarp];
    int root = rootbegin[iwarp];
    int lastroot = rootbegin[iwarp + 1];
    int firstnode = nodebegin[iwarp];
    int lastnode = nodebegin[iwarp + 1];

    triang_interleaved2_device(nt, ic, ncycle, stride, lastnode);
    bksub_interleaved2_device(nt, root + ic, lastroot, ic, ncycle, stride, firstnode);
}

void solve_interleaved2_launcher(NrnThread* nt, InterleaveInfo* info, int ncore, void* stream) {
    auto hip_stream = static_cast<hipStream_t>(stream);

    int threadsPerBlock = warpsize;
    // TODO: Should blocksPerGrid be a fixed number and have a while block inside the kernel?
    int blocksPerGrid = (ncore + threadsPerBlock - 1) / threadsPerBlock;

    solve_interleaved2_kernel<<<blocksPerGrid, threadsPerBlock, 0, hip_stream>>>(nt, info, ncore);

    hipStreamSynchronize(hip_stream);

    CHECKLAST("solve_interleaved2_launcher");
}

}  // namespace coreneuron
