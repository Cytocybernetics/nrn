#include "hip/hip_runtime.h"
/*
# =============================================================================
# Copyright (c) 2016 - 2021 Blue Brain Project/EPFL
#
# See top-level LICENSE file for details.
# =============================================================================.
*/
#include "coreneuron/utils/nrnmutdec.h"
#include "coreneuron/utils/randoms/nrnran123.h"

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include <mutex>

// In a GPU build this file will be compiled by NVCC as CUDA code
// In a CPU build this file will be compiled by a C++ compiler as C++ code
#ifdef __HIPCC__
#define CORENRN_MANAGED __managed__
#else
#define CORENRN_MANAGED
#endif

namespace {
/* global data structure per process */
CORENRN_MANAGED philox4x32_key_t g_k = {{0}};
OMP_Mutex g_instance_count_mutex;
std::size_t g_instance_count{};
constexpr double SHIFT32 = 1.0 / 4294967297.0; /* 1/(2^32 + 1) */
}  // namespace

namespace coreneuron {
std::size_t nrnran123_instance_count() {
    return g_instance_count;
}

#ifdef _OPENMP
static MUTDEC void nrnran123_mutconstruct() {
    if (!mut_) {
        MUTCONSTRUCT(1);
    }
}
#else
void nrnran123_mutconstruct() {}
#endif

/* if one sets the global, one should reset all the stream sequences. */
CORENRN_HOST_DEVICE uint32_t nrnran123_get_globalindex() {
    return g_k.v[0];
}

CORENRN_HOST_DEVICE void nrnran123_getseq(nrnran123_State* s, uint32_t* seq, char* which) {
    *seq = s->c.v[0];
    *which = s->which_;
}

CORENRN_HOST_DEVICE void nrnran123_setseq(nrnran123_State* s, uint32_t seq, char which) {
    if (which > 3) {
        s->which_ = 0;
    } else {
        s->which_ = which;
    }
    s->c.v[0] = seq;
    s->r = philox4x32(s->c, g_k);
}

CORENRN_HOST_DEVICE void nrnran123_getids(nrnran123_State* s, uint32_t* id1, uint32_t* id2) {
    *id1 = s->c.v[2];
    *id2 = s->c.v[3];
}

CORENRN_HOST_DEVICE void nrnran123_getids3(nrnran123_State* s,
                                           uint32_t* id1,
                                           uint32_t* id2,
                                           uint32_t* id3) {
    *id3 = s->c.v[1];
    *id1 = s->c.v[2];
    *id2 = s->c.v[3];
}

CORENRN_HOST_DEVICE uint32_t nrnran123_ipick(nrnran123_State* s) {
    uint32_t rval;
    char which = s->which_;
    rval = s->r.v[which++];
    if (which > 3) {
        which = 0;
        s->c.v[0]++;
        s->r = philox4x32(s->c, g_k);
    }
    s->which_ = which;
    return rval;
}

CORENRN_HOST_DEVICE double nrnran123_dblpick(nrnran123_State* s) {
    return nrnran123_uint2dbl(nrnran123_ipick(s));
}

CORENRN_HOST_DEVICE double nrnran123_negexp(nrnran123_State* s) {
    /* min 2.3283064e-10 to max 22.18071 */
    return -log(nrnran123_dblpick(s));
}

/* at cost of a cached  value we could compute two at a time. */
CORENRN_HOST_DEVICE double nrnran123_normal(nrnran123_State* s) {
    double w, x, y;
    double u1, u2;

    do {
        u1 = nrnran123_dblpick(s);
        u2 = nrnran123_dblpick(s);
        u1 = 2. * u1 - 1.;
        u2 = 2. * u2 - 1.;
        w = (u1 * u1) + (u2 * u2);
    } while (w > 1);

    y = sqrt((-2. * log(w)) / w);
    x = u1 * y;
    return x;
}

CORENRN_HOST_DEVICE double nrnran123_uint2dbl(uint32_t u) {
    /* 0 to 2^32-1 transforms to double value in open (0,1) interval */
    /* min 2.3283064e-10 to max (1 - 2.3283064e-10) */
    return ((double) u + 1.0) * SHIFT32;
}

/* nrn123 streams are created from cpu launcher routine */
void nrnran123_set_globalindex(uint32_t gix) {
    g_k.v[0] = gix;
}

nrnran123_State* nrnran123_newstream3(uint32_t id1,
                                      uint32_t id2,
                                      uint32_t id3,
                                      bool use_unified_memory) {
    nrnran123_State* s{nullptr};
    if (use_unified_memory) {
#ifdef __HIPCC__
        assert(hipMallocManaged(&s, sizeof(nrnran123_State)) == hipSuccess);
        assert(hipMemset(s, 0, sizeof(nrnran123_State)) == hipSuccess);
#else
        throw std::runtime_error("Tried to use CUDA unified memory in a non-GPU build.");
#endif
    } else {
        s = new nrnran123_State{};
    }
    s->c.v[0] = 0;
    s->c.v[1] = id3;
    s->c.v[2] = id1;
    s->c.v[3] = id2;
    nrnran123_setseq(s, 0, 0);
    {
        std::lock_guard<OMP_Mutex> _{g_instance_count_mutex};
        ++g_instance_count;
    }
    return s;
}

/* nrn123 streams are destroyed from cpu launcher routine */
void nrnran123_deletestream(nrnran123_State* s, bool use_unified_memory) {
    {
        std::lock_guard<OMP_Mutex> _{g_instance_count_mutex};
        --g_instance_count;
    }
    if (use_unified_memory) {
#ifdef __HIPCC__
        hipFree(s);
#else
        throw std::runtime_error("Tried to use CUDA unified memory in a non-GPU build.");
#endif
    } else {
        delete s;
    }
}
}  // namespace coreneuron
