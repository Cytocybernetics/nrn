/*
# =============================================================================
# Copyright (c) 2016 - 2021 Blue Brain Project/EPFL
#
# See top-level LICENSE file for details.
# =============================================================================
*/


#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <ostream>

void print_gpu_memory_usage() {
    size_t free_byte;
    size_t total_byte;

    hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);

    if (hipSuccess != cuda_status) {
        printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status));
        exit(1);
    }

    double free_db = (double) free_byte;
    double total_db = (double) total_byte;
    double used_db = total_db - free_db;
    printf(" GPU Memory (MBs) : Used = %f, Free = %f, Total = %f\n",
           used_db / 1024.0 / 1024.0,
           free_db / 1024.0 / 1024.0,
           total_db / 1024.0 / 1024.0);
    fflush(stdout);
}

void start_cuda_profile() {
    hipProfilerStart();
}

void stop_cuda_profile() {
    hipProfilerStop();
}
